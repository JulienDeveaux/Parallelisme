#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "util/util.h"

#include <stdio.h>
#include <stdlib.h>


#define N 5000
#define DIM 256			//taille de bloc impérativement en puissance de 2

//nvcc reduction.cu -o reduction
//./reduction

__global__ void sum(float *in, float *out, int n) {
	__shared__ float cache[DIM];
	int offset = blockIdx.x * blockDim.x; //début de la partie du tableau de ce bloc
	int i = threadIdx.x; //numéro du thread dans le bloc

	//copie du tableau dans la mémoire partagée
	if(offset + i < n) {
		cache[i] = in[offset + i];
	} else {
		cache[i] = 0;	// au cas ou on est a la fin du tableau ça change pas la somme
	}

	//réduction partielle dans la mémoire partagée
	int k = blockDim.x / 2;
	while(k > 0) {
		__syncthreads();	// a ne pas mettre dans le if sinon deadlock (tous les threads doivent le toucher)
		if(i < k) {
			cache[i] += cache[i + k];
		}
		k /= 2;
	}

	//copie du résultat dans la mémoire globale
	if(i == 0) {
		out[blockIdx.x] = cache[0];
	}
}

int main(int argc, char const *argv[]) {
	float a[N];
	float *dev_partial1, *dev_partial2, *tmp;

	for(int i = 0; i < N; i++) {
		a[i] = i + 1;
	}

	int n = N;
	int numBlocks = (n + DIM - 1) / DIM;
	HANDLE_CUDA_ERR(hipMalloc(&dev_partial1, n * sizeof(float)));
	HANDLE_CUDA_ERR(hipMalloc(&dev_partial2, numBlocks * sizeof(float)));
	HANDLE_CUDA_ERR(hipMemcpy(dev_partial1, a, n * sizeof(float), hipMemcpyHostToDevice));

	while(n > 1) {
		sum<<<numBlocks, DIM>>>(dev_partial1, dev_partial2, n);
		n = numBlocks;
		numBlocks = (n + DIM - 1) / DIM;
		tmp = dev_partial1;
		dev_partial1 = dev_partial2;
		dev_partial2 = tmp;
	}

	float sumGPU;
	HANDLE_CUDA_ERR(hipMemcpy(&sumGPU, dev_partial1, sizeof(float), hipMemcpyDeviceToHost));
	printf("Somme attendue : %.2f Somme obtenue : %.2f\n", N * (N + 1) / 2.0, sumGPU);

	HANDLE_CUDA_ERR(hipFree(dev_partial1));
	HANDLE_CUDA_ERR(hipFree(dev_partial2));
    return 0;
}