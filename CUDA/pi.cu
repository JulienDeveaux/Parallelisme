#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "util/util.h"

#include <stdio.h>
#include <stdlib.h>
// --- Timing includes
#include "GPUTiming/TimingCPU.h"
#include "GPUTiming/TimingGPU.cuh"
#include "GPUTiming/TimingCPU.cpp"
#include "GPUTiming/TimingGPU.cu"

#define N 1000000
#define DIM 256

__global__ void integrale(float *res, int n) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    float h = 1.0/n;
    
    float x1 = i*h;
    float x2 = x1 + h;
    
    float y1 = sqrt(1-x1*x1);
    float y2 = sqrt(1-x2*x2);
    
    if(i < n) res[i] = (y1+y2)*h / 2;

}

__global__ void sum(float *in, float *out, int n) {
    __shared__ float cache[DIM];
    int offset = blockIdx.x * blockDim.x; //début de la partie du tableau de ce bloc
    int i = threadIdx.x; //numéro du thread dans le bloc

    //copie du tableau dans la mémoire partagée
    if(offset + i < n) {
        cache[i] = in[offset + i];
    } else {
        cache[i] = 0;   // au cas ou on est a la fin du tableau ça change pas la somme
    }

    //réduction partielle dans la mémoire partagée
    int k = blockDim.x / 2;
    while(k > 0) {
        __syncthreads();    // a ne pas mettre dans le if sinon deadlock (tous les threads doivent le toucher)
        if(i < k) {
            cache[i] += cache[i + k];
        }
        k /= 2;
    }

    //copie du résultat dans la mémoire globale
    if(i == 0) {
        out[blockIdx.x] = cache[0];
    }
}

int main(int argc, char const *argv[]) {
    TimingGPU timer_GPU;
    timer_GPU.StartCounter();

    float total;
    float *dev_res, *dev_res2, *tmp;

    //Allouer la mémoire device
    hipMalloc(&dev_res, N * sizeof(float));
    hipMalloc(&dev_res2, N * sizeof(float));
    hipMalloc(&tmp, N * sizeof(float));
    
    //Kernel
    int numBlocks = (N+DIM-1) / DIM;
    integrale<<<numBlocks, DIM>>>(dev_res, N);

    int n = N;
    while(n > 1) {
        sum<<<numBlocks, DIM>>>(dev_res, dev_res2, n);
        n = numBlocks;
        numBlocks = (n + DIM - 1) / DIM;
        tmp = dev_res;
        dev_res = dev_res2;
        dev_res2 = tmp;
    }
    

    //Transfert de la mémoire vers host
    hipMemcpy(&total, dev_res, sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(dev_res);
    hipFree(dev_res2);

    printf("PI: %.9f\n\n", total*4);

    /*float res[N];
    float *dev_res;

    //Allouer la mémoire device
    hipMalloc(&dev_res, N * sizeof(float));
    
    //Kernel
    int numBlocks = (N+DIM-1) / DIM;
    integrale<<<numBlocks, DIM>>>(dev_res, N);
    
    //Transfert de la mémoire vers host
    hipMemcpy(res, dev_res, N*sizeof(float), hipMemcpyDeviceToHost);    
    
    hipFree(dev_res);
    
    //Resultats
    float total = 0.0;
    for(int i=0; i<N; i++) {total += res[i];}

    printf("PI: %.9f\n", total*4);*/

    printf("GPU Timing = %lf ms\n", timer_GPU.GetCounter());
    return 0;
}