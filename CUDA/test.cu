#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 1000000
#define DIM 256

__global__ void calc(float *x, float *y, float *res) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(x[index]*x[index]+y[index]*y[index] <= 1) {
        res[0]++;
    }
}

int main()
{
    printf("pi");
    float pi, x[N], y[N], res[1];
    float *dev_x, *dev_y, *dev_res;
    srand(time(NULL));

    for(int i = 0; i < N - 1; i++) {
        printf("%d\n", i);
        x[i] = rand() % 1;
        y[i] = rand() % 1;
    }

    hipMalloc(&dev_x, N*sizeof(float));
    hipMalloc(&dev_y, N*sizeof(float));
    hipMalloc(&dev_res, sizeof(float));

    hipMemcpy(dev_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    calc<<<N / DIM, DIM>>>(dev_x, dev_y, dev_res);

    hipMemcpy(res, &dev_res, sizeof(float), hipMemcpyDeviceToHost);

    pi = res[0]/N*4;
    printf("pi : %f\n", pi);

    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_res);

    return 0;
}//https://www.deleze.name/marcel/sec2/cours/CalculIntegral/pi_calcul.pdf