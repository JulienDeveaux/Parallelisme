#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/gpu_anim.h"

#define WIDTH 800
#define HEIGHT 600
#define DIM 16
#define M 200
#define K 3
#define G 28

//nvcc heat.cu -o heat -lglut -lGL -lGLU
//__NV_PRIME_RENDER_OFFLOAD=1 __GLX_VENDOR_LIBRARY_NAME=nvidia ./automate

__global__ void color(int *t, uchar4 *buf) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < WIDTH && y < HEIGHT) {
		int offset = y * WIDTH + x;
		
		float r, g, b;
		if(t[offset] > (M / 3) && t[offset] < (2*M / 3)) {	// M entre 1/3 et 2/3
			r = 100;
			g = 100;
			b = 100;
		} else if(t[offset] > (2*M / 3)) {					// M entre 2/3 et 3/3
			r = 200;
			g = 200;
			b = 200;
		} else {
			r = 0;
			g = 0;
			b = 0;
		}
		buf[offset].x = r;
		buf[offset].y = g;
		buf[offset].z = b;
		buf[offset].w = 255;		
	}
}

__global__ void diffuse(int *t_current, int *t_next) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < WIDTH && y < HEIGHT) {
		int offset = y * WIDTH + x;
		int top = y == HEIGHT - 1 ? offset : offset + WIDTH;
		int bottom = y == 0 ? offset : offset - WIDTH;
		int left = x == 0 ? offset : offset - 1;
		int right = x == WIDTH - 1 ? offset : offset + 1;
		int topRight = offset;
		int topLeft = offset;
		int bottomRight = offset;
		int bottomLeft = offset;
		if(y != HEIGHT && x != WIDTH) {
			topRight = offset + WIDTH + 1;
		}
		if(y != HEIGHT && x != 0) {
			topLeft = offset + WIDTH - 1;
		}
		if(y != 0 && x != WIDTH) {
			bottomRight = offset - WIDTH + 1;
		}
		if(y != 0 && x != 0) {
			bottomLeft = offset - WIDTH - 1;
		}

    	int nbNonSain = 0;
    	int moy = 0;
    	if(t_current[top] != 0) {
    		nbNonSain++;
    		moy += t_current[top];
    	}
    	if(t_current[bottom] != 0) {
    		nbNonSain++;
    		moy += t_current[bottom];
    	}
    	if(t_current[left] != 0) {
    		nbNonSain++;
    		moy += t_current[left];
    	}
    	if(t_current[right] != 0) {
    		nbNonSain++;
    		moy += t_current[right];
    	}
    	if(t_current[topRight] != 0) {
    		nbNonSain++;
    		moy += t_current[topRight];
    	}
    	if(t_current[topLeft] != 0) {
    		nbNonSain++;
    		moy += t_current[topLeft];
    	}
    	if(t_current[bottomLeft] != 0) {
    		nbNonSain++;
    		moy += t_current[bottomLeft];
    	}
    	if(t_current[bottomRight] != 0) {
    		nbNonSain++;
    		moy += t_current[bottomRight];
    	}


		if(t_current[offset] == M) { 					//si la cellule est malade elle devient saine

    		t_next[offset] = 0;

    	} else if (t_current[offset] == 0) {			//si elle est saine, (#voisins non sains)/K)

			t_next[offset] = nbNonSain / K;

    	} if (t_current[offset] != 0 && t_current[offset] != M) {	//si la cellule est infectée

    		t_next[offset] = ((t_current[offset]+moy) / (nbNonSain+1)) + G;

    	}

    	if(t_next[offset] > M) {
    		t_next[offset] = M;
    	}
	}
}

struct Data {
	int *t1;
	int *t2;
	dim3 blocks;
	dim3 threads;
};

void render_callback(uchar4 *buf, Data *data, int ticks) {
	diffuse<<<data->blocks, data->threads>>>(data->t1, data->t2);
	diffuse<<<data->blocks, data->threads>>>(data->t2, data->t1);
	color<<<data->blocks, data->threads>>>(data->t1, buf);
}

void clean_callback(Data *data) {
	HANDLE_ERROR(hipFree(data->t1));
	HANDLE_ERROR(hipFree(data->t2));
}

int main() {
	Data data;
	GPUAnimBitmap bitmap(WIDTH, HEIGHT, &data);
		
	size_t size = WIDTH * HEIGHT * sizeof(int);
	int *t_initial = (int *)calloc(WIDTH * HEIGHT, sizeof(int));

	srand(time(NULL));
	for(int i = 0; i < WIDTH * HEIGHT; i++) {
		int r = rand() % M+1;
		t_initial[i] = r;
	}
	
	data.blocks = dim3((WIDTH + DIM - 1) / DIM, (HEIGHT + DIM - 1) / DIM);
	data.threads = dim3(DIM, DIM);
	HANDLE_ERROR(hipMalloc(&data.t1, size));
	HANDLE_ERROR(hipMalloc(&data.t2, size));
	HANDLE_ERROR(hipMemcpy(data.t1, t_initial, size, hipMemcpyHostToDevice));
	
	bitmap.anim_and_exit((void (*)(uchar4*, void*, int))render_callback, (void (*)(void*))clean_callback);
	return 0;
}

