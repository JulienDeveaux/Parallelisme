#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "util/util.h"
#include "util/gpu_bitmap.h"
#include <GL/freeglut.h>

#include <stdio.h>
#include <stdlib.h>


#define N 5000
#define WIDTH 800
#define HEIGHT 600
#define DIM 16
#define K 1.0e-1f


//nvcc diffusion.cu -o -lglut -DREPORT_FPS -lGL diffusion
//./diffusion

struct Data {
	float *t1, *t2; // tableaux températures
	dim3 blocks, threads; // dimensions de la grille
};

__global__ void diffuse(float *t_current, float *t_next) { // un pas de temps
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x < WIDTH && y < HEIGHT) {
		int offset = y * WIDTH + x;
		int top = y == HEIGHT - 1 ? offset : offset + WIDTH;
		int bottom = y == 0 ? offset : offset - WIDTH;
		int left = x == 0 ? offset : offset - 1;
		int right =x == WIDTH - 1 ? offset : offset + 1;

		t_next[offset] = (1 - 4 * K) * t_current[offset] + K * (t_current[top] +
			t_current[bottom] + t_current[left] + t_current[right]);
	}
}

__global__ void color(float *t, uchar4 *buf) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x < WIDTH && y < HEIGHT) {
		int offset = y * WIDTH + x;
		float t3 = 3 * t[offset];
		float r, g, b;
		if(t3 < 1) {
			r = t3 * 255;
			g = 0;
			b = 0;
		} else if(t3 < 2) {
			r = 255;
			g = (t3 - 1) * 255;
			b = 0;
		} else {
			r = 255;
			g = 255;
			b = (t3 - 2) * 255;
		}
		buf[offset].x = r;
		buf[offset].y = g;
		buf[offset].z = b;
		buf[offset].w = 255;
	}
}

void render_callback(uchar4 *buf, Data *data, int ticks)  {
	diffuse<<<data->blocks, data->threads>>>(data->t1, data->t2);
	diffuse<<<data->blocks, data->threads>>>(data->t2, data->t1);
	color<<<data->blocks, data->threads>>>(data->t1, buf);
}

void clean_callback(Data *data) {
	HANDLE_CUDA_ERR(hipFree(data->t1));
	HANDLE_CUDA_ERR(hipFree(data->t2));
}

int main() {
	Data data;
	GPUBitmap bitmap(WIDTH, HEIGHT, &data, "Heat");

	size_t size = WIDTH * HEIGHT * sizeof(float);
	float *t_initial = (float*)malloc(size);
	//init la temp de chaque cellule

	//init data
	data.blocks = dim3((WIDTH + DIM - 1) / DIM, (HEIGHT + DIM - 1) / DIM);
	data.threads = dim3(DIM, DIM);
	HANDLE_CUDA_ERR(hipMalloc(&data.t1, size));
	HANDLE_CUDA_ERR(hipMalloc(&data.t2, size));
	HANDLE_CUDA_ERR(hipMemcpy(data.t1, t_initial, size, hipMemcpyHostToDevice));

	//lancer la boucle d'annimation
	bitmap.animate((void (*)(uchar4*, void*, int))render_callback, (void (*)(void*))clean_callback);
}